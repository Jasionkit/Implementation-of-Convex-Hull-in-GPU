#include "hip/hip_runtime.h"
#include "wb.h"
#include<bits/stdc++.h>
#include<vector>
#include<fstream>
#include<string.h>
#include<sstream>
#include<stdio.h>
#include<thrust/device_vector.h>
#include<thrust/copy.h>
#include<thrust/scan.h>
#include <thrust/sort.h>
#include<vector>
#include<climits>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>
#include<thrust/extrema.h>
#include<algorithm>
//#include<Timer.h>

#define CUDA_CHECK(ans)                                                   \
{ gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

using namespace std;

typedef pair<long int,long int> Point;

struct convexHull
{
   Point point;
   long int label;
   long int distance;
   int mark;
};

struct assignMax
{
   long int max;
   Point p;
   int l;
   int index;  
};

Point *hull;
Point *lhull;
long int inputLength,itr;
Point leftmost_point{INT_MAX,0},rightmost_point{INT_MIN,0};
convexHull *hostInput;
convexHull *deviceInput;
convexHull *original;
Point *deviceHull;
long int hull_length = 2;
long int lhull_length = 2;
convexHull *appendPoint;
long int append_point_len = 0;
assignMax *devMax;
assignMax *hostMax;
bool flag = false;
int maxlabel = 1;

bool comparision(Point a,Point b)
{
    return (a.first<b.first);
}

bool labelsort(convexHull a, convexHull b)
{
   return a.label<b.label;
}

/*
 @region kernel functions
*/


__global__ void calculate_perpendicularDistance_And_markNegDistance(convexHull *input,Point *devHull,long int size)
{
   long int Idx = threadIdx.x+blockIdx.x*blockDim.x;

   /*
   @ calculate perpendicular point
   */
   if((Idx)<size)
   {

     Point P = input[Idx].point;
     long int lb = input[Idx].label;
     Point min = devHull[lb];
     Point max = devHull[lb+1];
     input[Idx].distance = (P.second-min.second)*(max.first-min.first)-(max.second-min.second)*(P.first-min.first);

     if(input[Idx].distance<0)
     {
        input[Idx].mark = -1;
     }else
     {
        input[Idx].mark = 1;
     }
     
   }  
}

__global__ void scan(convexHull *input,assignMax *store,int size,bool upper)
{
   int Idx = blockIdx.x*blockDim.x+threadIdx.x;
   long int itr = 0;
   
   if(upper)
   {
     for(;itr<size;)
     {
        if(Idx==input[itr].label)
        {
          while(itr<size&&Idx==input[itr].label)
          {
             if((input[itr].distance>0)&&(store[Idx].max<input[itr].distance))
             {
                store[Idx].max = input[itr].distance;
                store[Idx].p.first = input[itr].point.first;
                store[Idx].p.second = input[itr].point.second;
                store[Idx].l = input[itr].label;
                store[Idx].index = itr;
             }
             itr++;
          }
          break;
        }
        itr++;
     }
    }
    else
    {
       for(;itr<size;)
       {
          if(Idx==input[itr].label)
          {
            while(itr<size&&Idx==input[itr].label)
            {
               if((input[itr].distance<0)&&(store[Idx].max>input[itr].distance))
               {
                  store[Idx].max = input[itr].distance;
                  store[Idx].p.first = input[itr].point.first;
                  store[Idx].p.second = input[itr].point.second;
                  store[Idx].l = input[itr].label;
                  store[Idx].index = itr;
               }
               itr++;
            }
            break;
          }
          itr++;
       }
    }
}

__global__ void update_label(convexHull *ptr, assignMax *M,long int size)
{
  int idx = threadIdx.x+blockIdx.x*blockDim.x;

  int l = M[idx].l;
  long int i=0;
  for(;i<size;i++)
  {
      if(l==ptr[i].label&&M[idx].p.first<=ptr[i].point.first)
      {
        ptr[i].label = ptr[i].label+1;
      }
  }

}

void initialize(convexHull ptr[],long int n)
{
   long int i=0;
   for(;i<n;i++)
   {
      if(leftmost_point.first>ptr[i].point.first)
      {
        leftmost_point = ptr[i].point;
      }
      if(rightmost_point.first<ptr[i].point.first)
      {
        rightmost_point = ptr[i].point;
      }
   }
}

void initialize_Max(assignMax ptr[],long int n,bool upper)
{
   long int i=0;
   for(;i<n;i++)
   {
     if(upper)
     {
       ptr[i].max = INT_MIN;
     }
     else
     {
       ptr[i].max = INT_MAX;
     }
     ptr[i].p = {INT_MIN,INT_MIN};
     ptr[i].l = -1;
     ptr[i].index = -1;
   }
}

void update_Hull(int labels,bool upper)
{
  flag = true;

  if(upper)
  {
      for(int k=0;k<labels;k++)
      {
        flag = true;
        for(long int hull_itr=0;hull_itr<hull_length;hull_itr++)
        {
            if((hostInput[hostMax[k].index].point.first==hull[hull_itr].first)&&(hostInput[hostMax[k].index].point.second==hull[hull_itr].second))
            {
               // for distinct point
               flag = false;
               break;
            }
        }

        if(flag&&hostMax[k].l!=-1)
        {
            hull[hull_length] = hostInput[hostMax[k].index].point;
            hull_length++;
        }
      }
   }else
   {
      for(int k=0;k<labels;k++)
      {
        flag = true;
        for(long int hull_itr=0;hull_itr<lhull_length;hull_itr++)
        {
            if((hostInput[hostMax[k].index].point.first==lhull[hull_itr].first)&&(hostInput[hostMax[k].index].point.second==lhull[hull_itr].second))
            {
               // for distinct point
               flag = false;
               break;
            }
        }

        if(flag&&hostMax[k].l!=-1)
        {
            lhull[lhull_length] = hostInput[hostMax[k].index].point;
            lhull_length++;
        }
      }
   }
}

void update_And_Remove_MarkPoints(convexHull p[],long int &p_len,convexHull ap[],long int &ap_len)
{
    long int i=0;
    for(long int k=0;k<p_len;k++)
    {
       if(p[k].mark==-1)
       {
         ap[ap_len].point.first = p[k].point.first;
         ap[ap_len].point.second = p[k].point.second;
         ap[ap_len].label = p[k].label;
         ap[ap_len].distance = p[k].distance;
         ap[ap_len].mark = p[k].mark;
         ap_len++;
       }else
       {
         p[i] = p[k];
         i++;
       }
    }
    p_len = i;
}

void printOutput()
{
   int it = 0;
   for(;it<hull_length;it++)
   {
      cout<<hull[it].first<<" "<<hull[it].second<<endl;
   }
}

int main(int argc, char *argv[]) {
  
  wbTime_start(Generic, "Importing data and creating memory on host");
  ifstream file;
  file.open(argv[2]);
  file>>inputLength;
  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  /*
  @region initialization
  */
  
  hostInput = new convexHull[inputLength];
  deviceInput = new convexHull[inputLength];
  original = new convexHull[inputLength];
  hull = new Point[inputLength];
  deviceHull = new Point[inputLength];
  appendPoint = new convexHull[inputLength];
  
  /*
  @endregion
  */
   
  //Assigining Value

  for(itr=0;itr<inputLength;itr++)
  {
     file>>hostInput[itr].point.first;
     file>>hostInput[itr].point.second;
     hostInput[itr].label = 0;
     original[itr].label = 0;
     original[itr].distance = 0;
     hostInput[itr].distance = 0;
     original[itr].point.first = hostInput[itr].point.first;
     original[itr].point.second = hostInput[itr].point.second;
  }
  file.close();

  int threads_per_block = 512;
  dim3 blocks(ceil(inputLength/threads_per_block)+1,1,1);
  

  //find the left and righ most point form region
  initialize(hostInput,inputLength);
  
  hull[0] = leftmost_point;
  hull[1] = rightmost_point;
  
  flag = false;
  
  //GpuTimer time;
  //time.Start();

  /*
   @params [upper convexHull]
   @descriptor {finding the point which is the part of lower hull}
  */
  
  bool lower_flag = false; 

  int prev_len = 2;
  
  clock_t t = clock();

  do
  {
      //compute upperhull
      
      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);

      hipMalloc((void **)&deviceHull,inputLength*sizeof(Point));
      hipMemcpy(deviceHull,hull,inputLength*sizeof(Point),hipMemcpyHostToDevice);

      calculate_perpendicularDistance_And_markNegDistance<<<blocks,threads_per_block>>>(deviceInput,deviceHull,inputLength);
      hipMemcpy(hostInput,deviceInput,inputLength*sizeof(convexHull),hipMemcpyDeviceToHost);
      
      //sort based on the label
      
      std::sort(hostInput,hostInput+inputLength,labelsort);
      
      int label_thread = hostInput[inputLength-1].label+1;

      devMax = new assignMax[label_thread];
      hostMax = new assignMax[label_thread];

      initialize_Max(hostMax,label_thread,true);
      
      hipMalloc((void **)&devMax,label_thread*sizeof(assignMax));
      hipMemcpy(devMax,hostMax,label_thread*sizeof(assignMax),hipMemcpyHostToDevice);

      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);
      
      scan<<<1,label_thread>>>(deviceInput,devMax,inputLength,true);
      hipMemcpy(hostMax,devMax,label_thread*sizeof(assignMax),hipMemcpyDeviceToHost);
      /*
       @method update hull
       @description [which have distinct points in hull]
      */

      prev_len = hull_length;

      update_Hull(label_thread,true);
      
      if(lower_flag==false){
        //update_And_Mark
        update_And_Remove_MarkPoints(hostInput,inputLength,appendPoint,append_point_len);
        lower_flag = true;
      }
      
      //sort label_partition
      
      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);

      update_label<<<1,label_thread>>>(deviceInput,devMax,inputLength);
      
      hipMemcpy(hostInput,deviceInput,sizeof(convexHull)*inputLength,hipMemcpyDeviceToHost);
      
      maxlabel = label_thread;

      std::sort(hull,hull+hull_length,comparision);
      
      hipDeviceSynchronize();
  
  }while(prev_len!=hull_length);
  
  //time.Stop();
  /*
  @params [upper hull]
  @descriptor {finding the point which is the part of upper hull, having -ve perpendicular distance}
  */
  
  appendPoint[append_point_len].point = leftmost_point;
  appendPoint[append_point_len].label = 0;
  append_point_len++;
  appendPoint[append_point_len].point = rightmost_point;
  appendPoint[append_point_len].label = 0;
  append_point_len++;
  inputLength = append_point_len;
  
  thrust::device_vector<convexHull> temp(appendPoint,appendPoint+append_point_len);
  thrust::copy(temp.begin(),temp.end(),hostInput);
  

  lhull = new Point[inputLength];

  lhull[0] = leftmost_point;
  lhull[1] = rightmost_point;

  // finding the lower hull
  
  maxlabel = 1;
  
  do
  {
      
      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);

      hipMalloc((void **)&deviceHull,inputLength*sizeof(Point));
      hipMemcpy(deviceHull,lhull,inputLength*sizeof(Point),hipMemcpyHostToDevice);

      calculate_perpendicularDistance_And_markNegDistance<<<blocks,threads_per_block>>>(deviceInput,deviceHull,inputLength);
      hipMemcpy(hostInput,deviceInput,inputLength*sizeof(convexHull),hipMemcpyDeviceToHost);
    

      //sort based on the label

      std::sort(hostInput,hostInput+inputLength,labelsort);//findlabel(inputLength)+1;
      
      int label_thread = hostInput[inputLength-1].label+1;

      devMax = new assignMax[label_thread];
      hostMax = new assignMax[label_thread];

      initialize_Max(hostMax,label_thread,false);

      hipMalloc((void **)&devMax,label_thread*sizeof(assignMax));
      hipMemcpy(devMax,hostMax,label_thread*sizeof(assignMax),hipMemcpyHostToDevice);

      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);
      
      scan<<<1,label_thread>>>(deviceInput,devMax,inputLength,false);
      hipMemcpy(hostMax,devMax,label_thread*sizeof(assignMax),hipMemcpyDeviceToHost);
      
      /*
       @method update hull
       @description [which have distinct points in hull]
      */
 
      prev_len = lhull_length;

      update_Hull(label_thread,false);
      
      //sort
      
      hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
      hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);

      update_label<<<1,label_thread>>>(deviceInput,devMax,inputLength);
      hipMemcpy(hostInput,deviceInput,sizeof(convexHull)*inputLength,hipMemcpyDeviceToHost);
      
      maxlabel = label_thread;

      std::sort(lhull,lhull+lhull_length,comparision);

      hipDeviceSynchronize();
  
  }while(prev_len!=lhull_length);
  
  t = clock() - t;

  cout<<"Total execution time is "<<(double)t/(double)CLOCKS_PER_SEC<<endl;
  /*
  @ param update the upperhull and lower hull
  */
  

  for(int j=0;j<lhull_length;j++)
  {
    bool check = true;
    for(int k=0;k<hull_length;k++){
       if(hull[k].first==lhull[j].first&&hull[k].second==lhull[j].second)
       {
         check = false;
         break;
       }
    }
    if(check)
    {
       hull[hull_length++] = lhull[j];
    }
  }

  printOutput();

  hipDeviceSynchronize();
  

  return 0;
}
