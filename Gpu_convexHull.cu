#include "hip/hip_runtime.h"
#include "wb.h"
#include<bits/stdc++.h>
#include<vector>
#include<fstream>
#include<string.h>
#include<sstream>
#include<stdio.h>

using namespace std;

typedef pair<long int,long int> Point;
Point *hull;

#define CUDA_CHECK(ans)                                                   \
{ gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}


/*
@ region Definition of global kernel function
*/

__global__ void lowerHull(Point *input, long int *label, long int *distance ,Point *devHull,long int size)
{
   long int Idx = blockIdx.x*blockDim.x+threadIdx.x;
   
   /*
   @ calculate perpendicular point
   */
   if((Idx)<size)
   {

     Point P = input[Idx];
     long lb = label[Idx];
     Point min = devHull[lb];
     Point max = devHull[lb+1];
     distance[Idx] = (P.second-min.second)*(max.first-min.first)-(max.second-min.second)*(P.first-min.first);
   }

}

__global__ void segmentedScan(Point *dist, Point *label, long int size)
{
    long int Idx = blockIdx.x*blockDim.x+threadIdx;
}

/*
@endregion
*/


int main(int argc, char *argv[]) {
  
  long int inputLength,itr;
  Point leftmost_point{INT_MAX,0},rightmost_point{INT_MIN,0};
  
  wbTime_start(Generic, "Importing data and creating memory on host");
  ifstream file;
  file.open(argv[2]);
  file>>inputLength;
  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  /*
  @region declaration
  */
  Point *hostInput = new Point[inputLength];
  Point *deviceInput = new Point[inputLength];;
  hull = new Point[inputLength];
  Point *deviceHull = new Point[inputLength];
  long int *Label = new long int[inputLength];
  long int *Distance = new long int[inputLength];
  long int *deviceLabel = new long int[inputLength];;
  long int *deviceDistance = new long int[inputLength];; 
  /*
  @endregion
  */

  memset(Label,0,inputLength*sizeof(long int));
  memset(Distance,0,inputLength*sizeof(long int));
 
  for(itr=0;itr<inputLength;itr++)
  {
     file>>hostInput[itr].first;
     file>>hostInput[itr].second;
  }
  file.close();

  int threads_per_block = 512;
  dim3 blocks(ceil(inputLength/threads_per_block)+1,1,1);
  
  /*
  @ param find the leftmost and rightmost Point in x direction
  */
  for(itr=0;itr<inputLength;itr++)
  {
     if(leftmost_point.first>hostInput[itr].first)
     {
        leftmost_point = hostInput[itr];
     }
     if(rightmost_point.first<hostInput[itr].first)
     {
        rightmost_point = hostInput[itr];
     }
  }

  /*
  @region Memory-Allocation
  */
  hipMalloc((void **)&deviceInput,inputLength*sizeof(Point));
  hipMemcpy(deviceInput,hostInput,inputLength*sizeof(Point),hipMemcpyHostToDevice);
  
  //--------------- insert point in hull -----------//
  hull[0] = leftmost_point;
  hull[1] = rightmost_point;
  

  hipMalloc((void **)&deviceHull,inputLength*sizeof(Point));
  hipMemcpy(deviceHull,hull,inputLength*sizeof(Point),hipMemcpyHostToDevice);
  hipMalloc((void **)&deviceLabel,inputLength*sizeof(long int));
  hipMemcpy(deviceLabel,Label,inputLength*sizeof(long int),hipMemcpyHostToDevice);
  hipMalloc((void **)&deviceDistance,inputLength*sizeof(long int));
  hipMemcpy(deviceDistance,Distance,inputLength*sizeof(long int),hipMemcpyHostToDevice); 
  
  /*
  @endregion
  */

  /*
  @ param calculate the LowerHull
  */
  
  lowerHull<<<blocks,threads_per_block>>>(deviceInput,deviceLabel,deviceDistance,deviceHull,inputLength);
  //upperHull<< blocks , threads_per_block >>( deviceInput, deviceLabel, deviceDistance, inputLength ); 
  //hipMemcpy(Distance,deviceDistance,inputLength*sizeof(Point),hipMemcpyDeviceToHost);
  
  segmentedScan<<<blocks , threads_per_block >>>(deviceDistance,deviceLabel,inputLength);
  
  /*
  @endregion
  */

  hipDeviceSynchronize();
  

  return 0;
}
