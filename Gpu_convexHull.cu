#include "hip/hip_runtime.h"
#include "wb.h"
#include<bits/stdc++.h>
#include<vector>
#include<fstream>
#include<string.h>
#include<sstream>
#include<stdio.h>
#include<thrust/device_vector.h>
#include<thrust/copy.h>
#include<thrust/scan.h>
#include <thrust/sort.h>
#include<vector>
#include<climits>

using namespace std;

typedef pair<long int,long int> Point;

struct convexHull
{
   Point point;
   long int label;
   long int distance;
   int mark;
};

struct assignMax
{
   long int max;
   long int index;  
};

Point *hull;

#define CUDA_CHECK(ans)                                                   \
{ gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}


/*
@ region Definition of global kernel function
*/

__global__ void lowerHull(convexHull *input,Point *devHull,long int size)
{
   long int Idx = blockIdx.x*blockDim.x+threadIdx.x;
   
   /*
   @ calculate perpendicular point
   */
   if((Idx)<size)
   {

     Point P = input[Idx].point;
     long int lb = input[Idx].label;
     Point min = devHull[lb];
     Point max = devHull[lb+1];
     input[Idx].distance = (P.second-min.second)*(max.first-min.first)-(max.second-min.second)*(P.first-min.first);
     if(input[Idx].distance<0)
     {
       input[Idx].mark = -1;
     }else
     {
       input[Idx].mark = 1;
     }
   }

}

__global__ void scan(convexHull *input,assignMax *store,int size)
{
   int Idx = blockIdx.x*blockDim.x+threadIdx.x;
   long int itr = 0;
  
   for(;itr<size;itr++)
   {
      if(Idx==input[itr].label)
      {
        while(itr<size&&Idx==input[itr].label)
        {
           if(store[Idx].max<input[itr].distance)
           {
              store[Idx].max = input[itr].distance;
              store[Idx].index = itr;
              //printf("%ld\n",store[Idx].max);
           }
           itr++;
        }
        break;
      }
   }
}


struct labelbased
{
  __host__ __device__ bool operator()(convexHull &x, convexHull &y)
  {
     return x.label<y.label;
  }
};

/*
@endregion
*/


int main(int argc, char *argv[]) {
  
  long int inputLength,itr;
  Point leftmost_point{INT_MAX,0},rightmost_point{INT_MIN,0};
  
  wbTime_start(Generic, "Importing data and creating memory on host");
  ifstream file;
  file.open(argv[2]);
  file>>inputLength;
  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  /*
  @region declaration
  */
  convexHull *hostInput = new convexHull[inputLength];
  convexHull *deviceInput = new convexHull[inputLength];
  convexHull *original = new convexHull[inputLength];
  hull = new Point[inputLength];
  Point *deviceHull = new Point[inputLength];
  long int hull_length = 2;
  /*
  @endregion
  */
 
  for(itr=0;itr<inputLength;itr++)
  {
     file>>hostInput[itr].point.first;
     file>>hostInput[itr].point.second;
     hostInput[itr].label = 0;
     original[itr].label = 0;
     original[itr].distance = 0;
     hostInput[itr].distance = 0;
     original[itr].point.first = hostInput[itr].point.first;
     original[itr].point.second = hostInput[itr].point.second;
  }
  file.close();

  int threads_per_block = 512;
  dim3 blocks(ceil(inputLength/threads_per_block)+1,1,1);
  
  /*
  @ param find the leftmost and rightmost Point in x direction
  */
  for(itr=0;itr<inputLength;itr++)
  {
     if(leftmost_point.first>hostInput[itr].point.first)
     {
        leftmost_point = hostInput[itr].point;
     }
     if(rightmost_point.first<hostInput[itr].point.first)
     {
        rightmost_point = hostInput[itr].point;
     }
  }

  /*
  @region Memory-Allocation
  */
  hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
  hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);
  
  //--------------- insert point in hull -----------//
  hull[0] = leftmost_point;
  hull[1] = rightmost_point;
  

  hipMalloc((void **)&deviceHull,inputLength*sizeof(Point));
  hipMemcpy(deviceHull,hull,inputLength*sizeof(Point),hipMemcpyHostToDevice);
  
  /*
  @endregion
  */

  /*
  @ param calculate the LowerHull
  */
  
  lowerHull<<<blocks,threads_per_block>>>(deviceInput,deviceHull,inputLength);
  hipMemcpy(hostInput,deviceInput,inputLength*sizeof(convexHull),hipMemcpyDeviceToHost);
  
  thrust::device_vector<convexHull> devI(hostInput,hostInput+inputLength);
  thrust::sort(devI.begin(),devI.end(),labelbased());
  thrust::copy(devI.begin(),devI.end(),hostInput);

  
  int label_thread = hostInput[inputLength-1].label+1;

  assignMax *devMax = new assignMax[label_thread];
  assignMax *hostMax = new assignMax[label_thread];
  for(int m=0;m<label_thread;m++)
  {
    hostMax[m].max = INT_MIN;
    hostMax[m].index = -1;
  }

  hipMalloc((void **)&devMax,label_thread*sizeof(assignMax));
  hipMemcpy(devMax,hostMax,label_thread*sizeof(assignMax),hipMemcpyHostToDevice);

  hipMalloc((void **)&deviceInput,inputLength*sizeof(convexHull));
  hipMemcpy(deviceInput,hostInput,inputLength*sizeof(convexHull),hipMemcpyHostToDevice);
  
  scan<<<1,label_thread>>>(deviceInput,devMax,inputLength);
  hipMemcpy(hostMax,devMax,label_thread*sizeof(assignMax),hipMemcpyDeviceToHost);  
  
  /*
   @method update hull
   @description []
  */
  
  for(int k=0;k<label_thread;k++)
  {
    hull[hull_length] = hostInput[hostMax[k].index].point;
    hull_length++;
  }
   
  /*
  @endregion
  */

  hipDeviceSynchronize();
  

  return 0;
}
