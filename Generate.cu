#include "hip/hip_runtime.h"
#include "wb.h"
#include<bits/stdc++.h>
#include<vector>
#include<fstream>
#include<string.h>
#include<sstream>
#include<stdio.h>
#include<thrust/device_vector.h>
#include<thrust/copy.h>
#include<thrust/scan.h>
#include <thrust/sort.h>
#include<vector>
#include<climits>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>
#include<thrust/extrema.h>

using namespace std;

typedef pair<long int,long int> Point;

set<Point> hull;


#define CUDA_CHECK(ans)                                                   \
{ gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

/*
***@ global function region
*/

__global__ void Launch_convexHull(Point *dPoints,long int psize,long int *dcheck,long int index,int *inc,int *dinc)
{
   int idx = blockIdx.x*blockDim.x+threadIdx.x;
   
   if(idx<psize){
   
     Point p = dPoints[index];
     Point q = dPoints[*inc];
     Point r = dPoints[dcheck[*dinc]];
     long int val = (q.second - p.second)*(r.first-q.first)-(q.first-p.first)*(r.second-q.second);
  
     __syncthreads();
     
     if(val<0)
     {

        atomicAdd(dinc,1);
        dcheck[*dinc] = *inc;
     }
     
     atomicAdd(inc,1);
     
     __syncthreads();
   
   }
}


int main(int argc, char *argv[]) {
  
  long int inputLength,itr;
  
  wbTime_start(Generic, "Importing data and creating memory on host");
  ifstream file;
  file.open(argv[2]);
  file>>inputLength;
  wbTime_stop(Generic, "Importing data and creating memory on host");
  
  /*
  @ declaration section
  */
  Point *hostPoints = new Point[inputLength];
  Point *devicePoints = new Point[inputLength];
  long int *check_orientation_point = new long int[inputLength];
  long int *device_check_orientation_point = new long int[inputLength];


  for(itr=0;itr<inputLength;itr++)
  {
     file>>hostPoints[itr].first;
     file>>hostPoints[itr].second;
  }
  file.close();
  /*
  @ region end
  */

  long int start = 0;
  for(itr=1;itr<inputLength;itr++)
  {
     if(hostPoints[itr].first<hostPoints[start].first)
     {
           start = itr;
     }
  }
  
  long int point = start,next;

  //Memory Allocation in GPU
  hipMalloc((void **)&devicePoints,sizeof(Point)*inputLength);
  hipMemcpy(devicePoints,hostPoints,sizeof(Point)*inputLength,hipMemcpyHostToDevice);
  
  int threads_per_block = 512;
  dim3 blocks(ceil(inputLength/threads_per_block)+1,1,1);
  
  int z=0;
  do
  {
      hull.insert(hostPoints[point]);
 
      next = (point+1)%inputLength;
      
      memset(check_orientation_point,-1,sizeof(long int)*inputLength);
      check_orientation_point[0] = next;

      hipMalloc((void **)&device_check_orientation_point,sizeof(long int)*inputLength);
      hipMemcpy(device_check_orientation_point,check_orientation_point,sizeof(long int)*inputLength,hipMemcpyHostToDevice);
      
      int *inc = (int *)malloc(sizeof(int));
      int *c_inc = (int *)malloc(sizeof(int));

      int *d_inc,*d_c_inc;
      hipMalloc((void **)&d_inc,sizeof(int));
      hipMemcpy(d_inc,inc,sizeof(int),hipMemcpyHostToDevice);
      hipMalloc((void **)&d_c_inc,sizeof(int));
      hipMemcpy(d_c_inc,c_inc,sizeof(int),hipMemcpyHostToDevice);


      Launch_convexHull<<<blocks,threads_per_block>>>(devicePoints,inputLength,device_check_orientation_point,point,d_inc,d_c_inc);
      
      hipMemcpy(check_orientation_point,device_check_orientation_point,sizeof(long int)*inputLength,hipMemcpyDeviceToHost);
      hipMemcpy(c_inc,d_c_inc,sizeof(int),hipMemcpyDeviceToHost); 
      hipMemcpy(inc,d_inc,sizeof(int),hipMemcpyDeviceToHost); 

      //do reduction
      
      point = check_orientation_point[*c_inc-1];

      z++;
  }while(z!=10);//while(point!=start);
  
  set<Point>::iterator it;
  
  for (it = hull.begin();it!=hull.end(); ++it)
  {
     cout<<it->first<<" "<<it->second<<endl;
  }

  hipDeviceSynchronize();
  

  return 0;
}